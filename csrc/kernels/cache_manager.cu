
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>

#include <algorithm>
#include <cassert>
#include <map>
#include <vector>

namespace vllm {

// Grid: (num_layers, num_pairs)
template <typename scalar_t>
__global__ void copy_blocks_kernel(int64_t* key_cache_ptrs,
                                   int64_t* value_cache_ptrs,
                                   const int64_t* __restrict__ block_mapping,
                                   const int64_t numel_per_block) {
    const int layer_idx = blockIdx.x;
    const int pair_idx = blockIdx.y;

    scalar_t* key_cache = reinterpret_cast<scalar_t*>(key_cache_ptrs[layer_idx]);
    scalar_t* value_cache =
        reinterpret_cast<scalar_t*>(value_cache_ptrs[layer_idx]);
    int64_t src_block_number = block_mapping[2 * pair_idx];
    int64_t dst_block_number = block_mapping[2 * pair_idx + 1];

    const int64_t src_block_offset = src_block_number * numel_per_block;
    const int64_t dst_block_offset = dst_block_number * numel_per_block;
    for (int i = threadIdx.x; i < numel_per_block; i += blockDim.x) {
        int64_t src_offset = src_block_offset + i;
        int64_t dst_offset = dst_block_offset + i;
        key_cache[dst_offset] = key_cache[src_offset];
    }
    for (int i = threadIdx.x; i < numel_per_block; i += blockDim.x) {
        int64_t src_offset = src_block_offset + i;
        int64_t dst_offset = dst_block_offset + i;
        value_cache[dst_offset] = value_cache[src_offset];
    }
}

}  // namespace vllm

// f16, bf16 are special cases: We use a 16-bit integer to simulate the bit width.
// SAFETY: This is technically UB due to aliasing, but it is OK because the width is compatible.
extern "C" {
void copy_blocks_f16(
    void* key_cache_ptrs,
    void* value_cache_ptrs,
    const void* block_mapping,
    int64_t num_layers,
    int64_t num_pairs,
    int64_t numel_per_block,
    hipStream_t stream) {
    dim3 grid(num_layers, num_pairs);
    dim3 block(std::min(int64_t(1024), numel_per_block));

    vllm::copy_blocks_kernel<int16_t><<<grid, block, 0, stream>>>(
        (int64_t*)key_cache_ptrs,
        (int64_t*)value_cache_ptrs,
        (const int64_t*)block_mapping,
        numel_per_block);
}
}

extern "C" {
void copy_blocks_bf16(
    void* key_cache_ptrs,
    void* value_cache_ptrs,
    const void* block_mapping,
    int64_t num_layers,
    int64_t num_pairs,
    int64_t numel_per_block,
    hipStream_t stream) {
    dim3 grid(num_layers, num_pairs);
    dim3 block(std::min(int64_t(1024), numel_per_block));

    vllm::copy_blocks_kernel<int16_t><<<grid, block, 0, stream>>>(
        (int64_t*)key_cache_ptrs,
        (int64_t*)value_cache_ptrs,
        (const int64_t*)block_mapping,
        numel_per_block);
}
}

namespace vllm {

template <typename scalar_t>
__global__ void reshape_and_cache_kernel(
    const scalar_t* __restrict__ key,          // [num_tokens, num_heads, head_size]
    const scalar_t* __restrict__ value,        // [num_tokens, num_heads, head_size]
    scalar_t* __restrict__ key_cache,          // [num_blocks, num_heads, head_size/x, block_size, x]
    scalar_t* __restrict__ value_cache,        // [num_blocks, num_heads, head_size, block_size]
    const int64_t* __restrict__ slot_mapping,  // [num_tokens]
    const int key_stride,
    const int value_stride,
    const int num_heads,
    const int head_size,
    const int block_size,
    const int x) {
    const int64_t token_idx = blockIdx.x;
    const int64_t slot_idx = slot_mapping[token_idx];
    if (slot_idx < 0) {
        // Padding token that should be ignored.
        return;
    }

    const int64_t block_idx = slot_idx / block_size;
    const int64_t block_offset = slot_idx % block_size;

    const int n = num_heads * head_size;
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        const int64_t src_key_idx = token_idx * key_stride + i;
        const int64_t src_value_idx = token_idx * value_stride + i;

        const int head_idx = i / head_size;
        const int head_offset = i % head_size;
        const int x_idx = head_offset / x;
        const int x_offset = head_offset % x;

        const int64_t tgt_key_idx = block_idx * num_heads * (head_size / x) * block_size * x + head_idx * (head_size / x) * block_size * x + x_idx * block_size * x + block_offset * x + x_offset;
        const int64_t tgt_value_idx = block_idx * num_heads * head_size * block_size + head_idx * head_size * block_size + head_offset * block_size + block_offset;
        key_cache[tgt_key_idx] = key[src_key_idx];
        value_cache[tgt_value_idx] = value[src_value_idx];
    }
}

#define CALL_RESHAPE_AND_CACHE(T)                                  \
    vllm::reshape_and_cache_kernel<T><<<grid, block, 0, stream>>>( \
        reinterpret_cast<T*>(key),                                 \
        reinterpret_cast<T*>(value),                               \
        reinterpret_cast<T*>(key_cache),                           \
        reinterpret_cast<T*>(value_cache),                         \
        slot_mapping,                                              \
        key_stride,                                                \
        value_stride,                                              \
        num_heads,                                                 \
        head_size,                                                 \
        block_size,                                                \
        x);

template <typename scalar_t>
__global__ void reshape_and_cache_flash_kernel(
    const scalar_t* __restrict__ key,          // [num_tokens, num_heads, head_size]
    const scalar_t* __restrict__ value,        // [num_tokens, num_heads, head_size]
    scalar_t* __restrict__ k_cache,            // [num_blocks, block_size, num_heads,
                                               // head_size]
    scalar_t* __restrict__ v_cache,            // [num_blocks, block_size, num_heads,
                                               // head_size]
    const int64_t* __restrict__ slot_mapping,  // [num_tokens]
    const int block_stride, const int key_stride, const int value_stride,
    const int num_heads, const int head_size, const int block_size) {
    const int64_t token_idx = blockIdx.x;
    const int64_t slot_idx = slot_mapping[token_idx];
    // NOTE: slot_idx can be -1 if the token is padded
    if (slot_idx < 0) {
        return;
    }
    const int64_t block_idx = slot_idx / block_size;
    const int64_t block_offset = slot_idx % block_size;
    const int n = num_heads * head_size;
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        const int64_t src_key_idx = token_idx * key_stride + i;
        const int64_t src_value_idx = token_idx * value_stride + i;
        const int head_idx = i / head_size;
        const int head_offset = i % head_size;
        const int64_t tgt_value_idx = block_idx * block_stride +
                                      block_offset * num_heads * head_size +
                                      head_idx * head_size + head_offset;
        k_cache[tgt_value_idx] = key[src_key_idx];
        v_cache[tgt_value_idx] = value[src_value_idx];
    }
}

#define CALL_RESHAPE_AND_CACHE_FLASH(T)                                  \
    vllm::reshape_and_cache_flash_kernel<T><<<grid, block, 0, stream>>>( \
        reinterpret_cast<T*>(key),                                       \
        reinterpret_cast<T*>(value),                                     \
        reinterpret_cast<T*>(key_cache),                                 \
        reinterpret_cast<T*>(value_cache),                               \
        slot_mapping,                                                    \
        block_stride,                                                    \
        key_stride,                                                      \
        value_stride,                                                    \
        num_heads,                                                       \
        head_size,                                                       \
        block_size);
}  // namespace vllm

extern "C" void reshape_and_cache(
    void* key,              // [num_tokens, num_heads, head_size]
    void* value,            // [num_tokens, num_heads, head_size]
    void* key_cache,        // [num_blocks, num_heads, head_size/x, block_size, x]
    void* value_cache,      // [num_blocks, num_heads, head_size, block_size]
    int64_t* slot_mapping,  // [num_tokens]

    int32_t num_tokens,
    int32_t num_heads,
    int32_t head_size,
    int32_t block_size,
    int32_t x,
    int32_t key_stride,
    int32_t value_stride,

    uint32_t dtype  // 0 => f16; 1 => bf16
) {
    dim3 grid(num_tokens);
    dim3 block(std::min(num_heads * head_size, 512));
    const hipStream_t stream = 0;

    if (dtype == 0) {
        CALL_RESHAPE_AND_CACHE(uint16_t);
    } else if (dtype == 1) {
        CALL_RESHAPE_AND_CACHE(__hip_bfloat16);
    }
}

extern "C" void reshape_and_cache_flash(
    void* key,              // [num_tokens, num_heads, head_size]
    void* value,            // [num_tokens, num_heads, head_size]
    void* key_cache,        // [num_blocks, num_heads, head_size, block_size]
    void* value_cache,      // [num_blocks, num_heads, head_size, block_size]
    int64_t* slot_mapping,  // [num_tokens]

    int64_t num_tokens,
    int64_t num_heads,
    int64_t head_size,
    int64_t block_size,
    int64_t key_stride,
    int64_t value_stride,
    int64_t block_stride,

    uint32_t dtype  // 0 => f16; 1 => bf16
) {
    dim3 grid(num_tokens);
    dim3 block(std::min(num_heads * head_size, int64_t(512)));
    const hipStream_t stream = 0;

    if (dtype == 0) {
        CALL_RESHAPE_AND_CACHE_FLASH(uint16_t);
    } else if (dtype == 1) {
        CALL_RESHAPE_AND_CACHE_FLASH(__hip_bfloat16);
    }
}
